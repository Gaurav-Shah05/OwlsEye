#include "hip/hip_runtime.h"
#include "quant_kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define FP16_LIMB_SIZE 16
#define FP32_LIMB_SIZE 32
#define FP16_TYPE uint16_t
#define FP32_TYPE uint32_t //**edited

// defining int32_constants as an array containing 16 elements of type uint32_t
__constant__ uint32_t int32_constants[16];
// same for this case
__constant__ uint64_t int64_constants[2];

// table lookup data, normally we use 5 bit (32 values). But this array size needs to be hardcoded before compilation.
// Use 128 to support up to 7 bits usigned table lookup.(the lookup assume 1 bit automaticcally used for sign => max 8 bits supported by this table)
__constant__ float table_lookup[128];
__constant__ float rounding_hint[128];

#define SIGN_MASK 0x80000000                     // 8000 => 1000 0000 0000 0000(16 bits)
#define FLOAT_SIGN_MASK 0x80000000           // => 1000 0000 0000 0000 0000 0000 0000 0000 (32 bits)
#define FLOAT_SIGN_RESET_MASK 0x7FFFFFFF     // => 1 0 bit 31 1 bits
#define SECOND_BIT_MASK 0x40000000               // 4000 => 0100 0000 0000 0000
#define FIXED_POSIT_INF 0x00000000               // defining the inf for posit
#define FIXED_POSIT_LIMB_ALL_BITS_SET 0xffffffff // all bits set for a posit
#define SINGLE_PRECISION_BIAS 127            // bias for the single precision number
#define FLOAT_SIZE 32                        // single precision
#define FLOAT_EXPONENT_MASK 0x7f800000       //=> 0111 1111 1000 0000 0000 0000 0000 0000
#define FLOAT_FRACTION_MASK 0x007fffff       //=> 0000 0000 0111 1111 1111 1111 1111 1111
#define FLOAT_SIGN_SHIFT 31
#define FLOAT_EXPONENT_SHIFT 23
#define FLOAT_DENORMAL_EXPONENT -126
#define FLOAT_HIDDEN_BIT_SET_MASK 0x00800000   //=> 0000 0000 1000 0000 0000 0000 0000 0000
#define FLOAT_SIGN_PLUS_EXP_LENGTH_MINUS_ONE 8 // name defined
#define TEMP_TYPE uint64_t                     // defining the temporary type to be unint64_t
#define UNSIGNED_LONG_LONG_SIZE 64             // name defined
#define EDP_ACC_SIZE 63                        // setting the accumulator size
#define FIXED_POSIT_EXP_SHIFT 41               // 64-23
#define FLOAT_EXP_SIGN_SHIFT 30
#define FLOAT_INF 0x7F800000         // 0111 1111 1000 0000 0000 0000 0000 0000
#define FLOAT_SIGN_PLUS_EXP_LENGTH 9 // name defined
#define FIXED_POSIT_LENGTH_PLUS_ONE 33 /

#define GET_MAX(a, b)           \
    ({                          \
        __typeof__(a) _a = (a); \
        __typeof__(b) _b = (b); \
        _a > _b ? _a : _b;      \
    })

#define _G_INFP 2147483648

union Bits
{
    float f;
    int32_t si;
    uint32_t ui;
};
typedef FP16_TYPE fp16;
typedef FP32_TYPE fp32;
#define _G_FPOSIT_SHIFT_AMOUNT int32_constants[0]
#define _G_MAXREALFP int32_constants[1]
#define _G_MINREALFP int32_constants[2]
#define FPOSIT_EXTRA_BITS_SHIFT int32_constants[3]
#define _G_USEED int32_constants[4]
#define _G_USEED_ZEROS int32_constants[5]
#define FPOSIT_EXPONENT_MASK int32_constants[6]

#define _G_MAXREAL_INT int32_constants[7]
#define _G_MINREAL_INT int32_constants[8]
#define _G_NBITS int32_constants[9]
#define _G_ESIZE int32_constants[10]

#define _G_RSIZE int32_constants[11]
#define FPOSIT_REGIME_MASK int32_constants[12]
#define FPOSIT_FRACTION_MASK int32_constants[13]
#define _G_FSIZE int32_constants[14]
#define _FP_REGIME_BIAS int32_constants[15]
// define regime size

#define FPOSIT_EXTRA_BITS_MASK int64_constants[0]
#define FPOSIT_HALFWAY_BIT_MASK int64_constants[1]

void generate_fixedposit_constants(int nsize, int es, int rf, uint32_t *int32_constants, uint64_t *int64_constants)
{
    // local vars have the same name as global constant vars, confusing but less likely error can happen here.
    // ugly but it's the straightforward conversion from the original #define macroes;
    // todo: make this one less messy

    _G_NBITS = nsize;
    _G_ESIZE = es;
    _G_RSIZE = rf;
    _G_FSIZE = nsize - es - rf - 1;

    if (nsize <= 32 && rf <= 10)
    {
        _G_FPOSIT_SHIFT_AMOUNT = FP32_LIMB_SIZE - nsize;
        // _G_MAXREALFP = ((1 << (nsize - 1)) - 1) << _G_FPOSIT_SHIFT_AMOUNT;
        _G_MAXREALFP = ((((1<<_G_RSIZE)-2)<<_G_ESIZE)|((1<<_G_ESIZE)-1))<<(_G_FSIZE+_G_FPOSIT_SHIFT_AMOUNT);
        // _G_MINREALFP = 1<<_G_FPOSIT_SHIFT_AMOUNT;
        _G_MINREALFP = 1<<(_G_ESIZE+_G_FSIZE+_G_FPOSIT_SHIFT_AMOUNT);
        FPOSIT_EXTRA_BITS_SHIFT = UNSIGNED_LONG_LONG_SIZE - nsize + 1;
        _G_USEED = 1 << (1 << es);
        _G_USEED_ZEROS = (1 << es);
        FPOSIT_EXPONENT_MASK = _G_USEED_ZEROS - 1;
        _FP_REGIME_BIAS = 0;

        _G_MAXREAL_INT = ((((rf-2)<<es)+((1<<es)-1))+SINGLE_PRECISION_BIAS)<<FLOAT_EXPONENT_SHIFT;
        // _G_MAXREAL_INT = ((_G_USEED_ZEROS * (nsize - 2)) + SINGLE_PRECISION_BIAS) << FLOAT_EXPONENT_SHIFT;
        // this is the maximum float integer that can be represented by the fixed posit.
        _G_MINREAL_INT = ((-1* (_G_RSIZE-1) * (_G_USEED_ZEROS))+SINGLE_PRECISION_BIAS) << FLOAT_EXPONENT_SHIFT;
        // _G_MINREAL_INT = ((_G_USEED_ZEROS * (2 - nsize)) + SINGLE_PRECISION_BIAS) << FLOAT_EXPONENT_SHIFT;
        // this is the minimum float integer that can be represented by the fixed posit.
        FPOSIT_REGIME_MASK = ((1 << rf) - 1) << (_G_FSIZE + _G_ESIZE);
        FPOSIT_FRACTION_MASK = ((1 << _G_FSIZE) - 1);

        FPOSIT_EXTRA_BITS_MASK = (1UL << (UNSIGNED_LONG_LONG_SIZE - nsize)) - 1;
        FPOSIT_HALFWAY_BIT_MASK = 1UL << (UNSIGNED_LONG_LONG_SIZE - nsize);
    }
    else
    {
        printf("unexpected fposit config\n");
        exit(1);
    }
};

__device__ __inline__ float fixedp16tofp32_gpu(fp32 Fp)
{
  union Bits v;
  v.si = 0;
	// get sign
	bool sign = Fp & SIGN_MASK;//
	Fp = (Fp ^ -sign) + sign;//
  // printf("the current float formed is: %d \n", v.si);
	// get the regime sign
	bool regime_sign = Fp & SECOND_BIT_MASK;//

	// get regime
	v.ui = Fp << 1;
	//int regime_length = (__builtin_clz(v.ui) & -!regime_sign) + (__builtin_clz(~v.ui) & -regime_sign);
  // printf("the current float formed is: %d \n", v.si);
	int m;
	  if(regime_sign)
	    m = (__builtin_clz(~v.ui));
	  else
	    m = (__builtin_clz(v.ui));
    // int regime_length = _G_RSIZE;
	int k2es = (m - regime_sign) << _G_ESIZE;
  // printf("the m is: %d \n", m);
	k2es = (k2es ^ -regime_sign) + regime_sign;
  // printf("the current float formed is: %d \n", v.si);
	// assemble
	v.ui <<= (_G_RSIZE);
	v.ui >>= (FLOAT_SIGN_PLUS_EXP_LENGTH - _G_ESIZE);
	// printf("the k2es is: %d \n", k2es);
  v.ui += ((SINGLE_PRECISION_BIAS - k2es) << FLOAT_EXPONENT_SHIFT);
  // printf("the current float formed is: %d \n", v.si);
	v.si ^= (FLOAT_INF ^ v.si) & -(Fp == _G_INFP);
	v.si ^= (0 ^ v.si) & -(Fp == 0);

	v.ui |= (sign << FLOAT_SIGN_SHIFT);
	return v.f;
}

__device__ __inline__ fp32 fp32tofixedp16_gpu(float f)
{
  fp32 Fp = 0;
  union Bits v;
  v.f = f;
  bool sign = v.ui & FLOAT_SIGN_MASK;
  v.ui &= 0x7FFFFFFF;
  // printf("the current input value is: %d \n", v.ui);
#ifdef FLOAT_ROUNDING
	uint16_t roundSign = sign << 15;
	if(v.ui > _G_MAXREAL_INT)
		return _G_INFP | roundSign;
	if(v.ui < _G_MINREAL_INT)
		return 0;
#endif
  // printf("the current output value is: %d \n", Fp);
  Fp ^= (Fp ^_G_MAXREALFP) & -(v.si >= _G_MAXREAL_INT);
  // printf("the current output value is: %d \n", Fp);
  Fp ^= (Fp ^ _G_INFP) & -(v.si >= FLOAT_INF);
  // printf("the current output value is: %d \n", Fp);
  Fp ^= (Fp ^ _G_MINREALFP) & -(v.si != 0 && v.si <= _G_MINREAL_INT);
  // printf("the current output value is: %d \n", Fp);
  // min posit exponent in 16, 3 is 112
  // therefore all the float subnormals will be handled
  // in the previous if statement

  // get exponent sign
  bool exp_sign = !(v.ui >> FLOAT_EXP_SIGN_SHIFT);

  //get regime and exponent
  uint32_t exp = abs((v.si >> FLOAT_EXPONENT_SHIFT) - SINGLE_PRECISION_BIAS);
  // printf("the current exponent value is: %d \n", exp);
  TEMP_TYPE regime_and_exp = (((1 << ((exp >> _G_ESIZE) + 1)) - 1) << (_G_RSIZE-((exp >> _G_ESIZE) + 1)+_G_ESIZE)) | (exp & FPOSIT_EXPONENT_MASK);
  //if exponent is negative
  // printf("the current regime and exp value is: %d \n", regime_and_exp);
  regime_and_exp = ((regime_and_exp ^ -exp_sign) + exp_sign);// >> ((exp_sign & !((exp & FPOSIT_EXPONENT_MASK))) & (bool) exp);
  // printf("the current regime and exp value is: %d \n", regime_and_exp);
  // int regime_and_exp_length = (exp >> _G_ESIZE) + 2 + _G_ESIZE - ((exp_sign & !((exp & FPOSIT_EXPONENT_MASK))) & (bool) exp);
  int regime_and_exp_length = _G_ESIZE+_G_RSIZE;

  //assemble
  regime_and_exp <<= (UNSIGNED_LONG_LONG_SIZE - regime_and_exp_length);
  regime_and_exp |= ((TEMP_TYPE) (v.ui & FLOAT_FRACTION_MASK) << (FIXED_POSIT_EXP_SHIFT - regime_and_exp_length));
  fp32 temp_p = (regime_and_exp >> FPOSIT_EXTRA_BITS_SHIFT);

  //round
  temp_p += (bool) (regime_and_exp & FPOSIT_HALFWAY_BIT_MASK) && ((temp_p & 1) | (regime_and_exp & FPOSIT_EXTRA_BITS_MASK));
  // printf("the current output value is: %d \n", Fp);
 if (_G_NBITS != 32)
    // {printf("the current output value is: %d \n", Fp);
    temp_p <<= _G_FPOSIT_SHIFT_AMOUNT;
    // }
    
  Fp ^= (temp_p ^ Fp) & -((v.si < _G_MAXREAL_INT) & (v.si > _G_MINREAL_INT));

  Fp = (Fp ^ -sign) + sign;
  // printf("the final output value is: %d \n", Fp);
  return Fp;
}


__global__ void fixed_posit_kernel_nearest(float *input, float *output, float scale, size_t input_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        float temp_input = input[index] * scale;

        fp32 temp = fp32tofixedp16_gpu(temp_input);
        temp_input = fixedp16tofp32_gpu(temp);

        output[index] = temp_input / scale;
    }
}

// __device__ float new_format_quantize_nearest(float input)
// {
//     float constants[32] = {1.0 / 65536, 1.0 / 32768, 1.0 / 16384, 1.0 / 8192, 1.0 / 4096, 1.0 / 2048, 1.0 / 1024, 1.0 / 512, 1.0 / 256, 1.0 / 128,
//                            3.0 / 256, 1.0 / 64, 5.0 / 256, 3.0 / 128, 7.0 / 256, 1.0 / 32, 9.0 / 256, 5.0 / 128, 3.0 / 64, 7.0 / 128,
//                            1.0 / 16, 9.0 / 128, 5.0 / 64, 3.0 / 32, 7.0 / 64, 1.0 / 8, 9.0 / 64, 3.0 / 16, 1.0 / 4, 3.0 / 8, 1.0 / 2, 1.0};
//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < 32; i++)
//         {
//             float abs_err = fabsf(constants[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = constants[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float act_format_quantize_nearest(float input)
// {

//     float constants[32] = {1.0 / 4096, 1.0 / 2048, 1.0 / 1024, 1.0 / 512, 1.0 / 256, 1.0 / 128, 1.0 / 64, 1.0 / 32, 1.0 / 16, 1.0 / 8, 3.0 / 16,
//                            1.0 / 4, 5.0 / 16, 3.0 / 8, 7.0 / 16, 1.0 / 2, 9.0 / 16, 5.0 / 8, 3.0 / 4, 7.0 / 8, 1.0, 9.0 / 8, 5.0 / 4, 3.0 / 2,
//                            7.0 / 4, 2.0, 9.0 / 4, 3.0, 4.0, 6.0, 8.0, 16.0};
//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < 32; i++)
//         {
//             float abs_err = fabsf(constants[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = constants[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float configurable_table_quantize_nearest(float input, int table_size)
// {

//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < table_size; i++)
//         {
//             float abs_err = fabsf(table_lookup[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = table_lookup[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float configurable_table_quantize_rounding_hint(float input, int table_size)
// {

//     float result = 0.0;
//     if (input != 0.0)
//     {
//         float min_constant = 0.0;
//         for (int i = 0; i < table_size; i++)
//         {
//             // float abs_err = fabs(constants[i] - fabs(input));
//             if (fabsf(input) > rounding_hint[i])
//                 min_constant = table_lookup[i];
//             // printf( " %f %f %f \n", input , rounding_hint[i] , min_constant);
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// // template <typename scalar_t>
// __global__ void newformat_kernel_nearest(float *input, float *output, float scale, size_t input_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = new_format_quantize_nearest(temp_input);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void actformat_kernel_nearest(float *input, float *output, float scale, size_t input_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = act_format_quantize_nearest(temp_input);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void configurable_table_kernel_nearest(float *input, float *output, float scale, size_t input_size, size_t table_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = configurable_table_quantize_nearest(temp_input, table_size);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void configurable_quantize_kernel_rounding_hint(float *input, float *output, float scale, size_t input_size, size_t table_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = configurable_table_quantize_rounding_hint(temp_input, table_size);

//         output[index] = temp_input / scale;
//     }
// }


void fixed_posit_kernel_nearest_wrapper(float *__restrict__ a,
                                  float *o, int size, int nsize, int es, int rf, float scale, int blockNums, int blockSize)
{

    uint32_t int32_constants_host[16];
    uint64_t int64_constants_host[2];
    generate_fixedposit_constants(nsize, es, rf, int32_constants_host, int64_constants_host);

    hipMemcpyToSymbol(HIP_SYMBOL(int32_constants), &int32_constants_host[0], 16 * sizeof(uint32_t), 0);
    hipMemcpyToSymbol(HIP_SYMBOL(int64_constants), &int64_constants_host[0], 2 * sizeof(uint64_t), 0);

    fixed_posit_kernel_nearest<<<blockNums, blockSize>>>(a,
                                                   o,
                                                   scale,
                                                   size);
}

// void newformat_kernel_nearest_wrapper(float *__restrict__ a,
//                                       float *o, int size, float scale, int blockNums, int blockSize)
// {

//     newformat_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                        o,
//                                                        scale,
//                                                        size);
// }

// void actformat_kernel_nearest_wrapper(float *__restrict__ a,
//                                       float *o, int size, float scale, int blockNums, int blockSize)
// {

//     actformat_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                        o,
//                                                        scale,
//                                                        size);
// }

// void configurable_quantize_kernel_nearest_wrapper(float *__restrict__ a,
//                                                   float *o,
//                                                   /*table lookup data*/
//                                                   float *constants,
//                                                   int table_size, int size, float scale, int blockNums, int blockSize)
// {

//     hipMemcpyToSymbol(HIP_SYMBOL(table_lookup), &constants[0], table_size * sizeof(float), 0);
//     configurable_table_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                                 o,
//                                                                 scale,
//                                                                 size,
//                                                                 table_size);
// }

// void configurable_quantize_kernel_rounding_hint_wrapper(float *__restrict__ a,
//                                                         float *o,
//                                                         /*table lookup data*/
//                                                         float *constants,
//                                                         float *round_hints,
//                                                         int table_size, int size, float scale, int blockNums, int blockSize)
// {

//     hipMemcpyToSymbol(HIP_SYMBOL(table_lookup), &constants[0], table_size * sizeof(float), 0);
//     hipMemcpyToSymbol(HIP_SYMBOL(rounding_hint), &round_hints[0], table_size * sizeof(float), 0);

//     configurable_quantize_kernel_rounding_hint<<<blockNums, blockSize>>>(a,
//                                                                          o,
//                                                                          scale,
//                                                                          size,
//                                                                          table_size);
// }